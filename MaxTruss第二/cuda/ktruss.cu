#include "hip/hip_runtime.h"
#include <thrust/count.h>
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/scan.h>

#include <algorithm>
#include <cstdlib>
#include <vector>

#include "log.h"
#include "util.h"

// 扫描支持边是否与truss层次相同
__global__ void ScanKernel(EdgeT halfEdgesNum, const NodeT *edgesSup, NodeT level, EdgeT *curr, EdgeT *currTail,
                           bool *inCurr) {
  auto from = blockDim.x * blockIdx.x + threadIdx.x;
  auto step = gridDim.x * blockDim.x;
  for (EdgeT i = from; i < halfEdgesNum; i += step) {
    if (edgesSup[i] == level) {
      inCurr[i] = true;
      curr[atomicAdd(currTail, 1)] = i;
    }
  }
}

// 扫描支持边层次小于指定层次
__global__ void ScanLessThanLevelKernel(EdgeT halfEdgesNum, const NodeT *edgesSup, NodeT level, EdgeT *curr,
                                        EdgeT *currTail, bool *inCurr) {
  auto from = blockDim.x * blockIdx.x + threadIdx.x;
  auto step = gridDim.x * blockDim.x;
  for (EdgeT i = from; i < halfEdgesNum; i += step) {
    if (edgesSup[i] <= level) {
      inCurr[i] = true;
      curr[atomicAdd(currTail, 1)] = i;
    }
  }
}

// 更新支持边的数值
__inline__ __device__ void UpdateSup(EdgeT e, NodeT *edgesSup, NodeT level, EdgeT *next, bool *inNext,
                                     EdgeT *nextTail) {
  NodeT supE = atomicSub(&edgesSup[e], 1);
  if (supE == (level + 1)) {
    auto insertIdx = atomicAdd(nextTail, 1);
    next[insertIdx] = e;
    inNext[e] = true;
  }
  if (supE <= level) {
    atomicAdd(&edgesSup[e], 1);
  }
}

__global__ void UpdateProcessKernel(const EdgeT *curr, const EdgeT currTail, bool *inCurr, bool *processed) {
  auto from = blockDim.x * blockIdx.x + threadIdx.x;
  auto step = gridDim.x * blockDim.x;
  for (EdgeT i = from; i < currTail; i += step) {
    EdgeT e = curr[i];
    processed[e] = true;
    inCurr[e] = false;
  }
}

template <typename T>
__host__ __device__ void swap(T &a, T &b) {
  T temp = a;
  a = b;
  b = temp;
}

__device__ EdgeT BinarySearch(NodeT target, const NodeT *adj, EdgeT start, EdgeT end) {
  EdgeT last = end;
  while (start < end) {
    EdgeT mid = start + ((end - start) >> 1u);
    if (adj[mid] < target) {
      start = mid + 1;
    } else if (adj[mid] > target) {
      end = mid;
    } else {
      return mid;
    }
  }
  return last;
}

__inline__ __device__ void PeelTriangle(NodeT level, const bool *inCurr, EdgeT *next, EdgeT *nextTail, bool *inNext,
                                        NodeT *edgesSup, bool *processed, EdgeT ee1, EdgeT ee2, EdgeT ee3) {
  if (processed[ee2] || processed[ee3]) {
    return;
  }
  if (edgesSup[ee2] > level && edgesSup[ee3] > level) {
    UpdateSup(ee2, edgesSup, level, next, inNext, nextTail);
    UpdateSup(ee3, edgesSup, level, next, inNext, nextTail);
  } else if (edgesSup[ee2] > level) {
    if ((ee1 < ee3 && inCurr[ee3]) || !inCurr[ee3]) {
      UpdateSup(ee2, edgesSup, level, next, inNext, nextTail);
    }
  } else if (edgesSup[ee3] > level) {
    if ((ee1 < ee2 && inCurr[ee2]) || !inCurr[ee2]) {
      UpdateSup(ee3, edgesSup, level, next, inNext, nextTail);
    }
  }
}

__global__ void SubLevelKernel(const EdgeT *nodeIndex, const NodeT *adj, const EdgeT *curr, bool *inCurr,
                               EdgeT currTail, NodeT *edgesSup, NodeT level, EdgeT *next, bool *inNext, EdgeT *nextTail,
                               bool *processed, const EdgeT *edgesId, const uint64_t *halfEdges) {
  __shared__ EdgeT size;
  extern __shared__ EdgeT shared[];
  EdgeT *eArr1 = shared;
  EdgeT *eArr2 = shared + blockDim.x * 2;
  EdgeT *eArr3 = shared + blockDim.x * 2 * 2;
  if (threadIdx.x == 0) {
    size = 0;
  }
  __syncthreads();

  for (EdgeT i = blockIdx.x; i < currTail; i += gridDim.x) {
    EdgeT e1 = curr[i];
    NodeT u = FIRST(halfEdges[e1]);
    NodeT v = SECOND(halfEdges[e1]);

    EdgeT uStart = nodeIndex[u];
    EdgeT uEnd = nodeIndex[u + 1];
    EdgeT vStart = nodeIndex[v];
    EdgeT vEnd = nodeIndex[v + 1];

    if (uEnd - uStart > vEnd - vStart) {
      swap(u, v);
      swap(uStart, vStart);
      swap(uEnd, vEnd);
    }

    for (auto e2 = uStart + threadIdx.x; e2 < uStart + DIV_ROUND_UP(uEnd - uStart, blockDim.x) * blockDim.x;
         e2 += blockDim.x) {
      __syncthreads();

      if (size >= blockDim.x) {
        for (EdgeT j = threadIdx.x; j < size; j += blockDim.x) {
          EdgeT ee1 = eArr1[j];
          EdgeT ee2 = edgesId[eArr2[j]];
          EdgeT ee3 = edgesId[eArr3[j]];
          PeelTriangle(level, inCurr, next, nextTail, inNext, edgesSup, processed, ee1, ee2, ee3);
        }
        __syncthreads();
        if (threadIdx.x == 0) {
          size = 0;
        }
        __syncthreads();
      }

      EdgeT e3 = vEnd;
      if (e2 < uEnd) {
        e3 = BinarySearch(adj[e2], adj, vStart, vEnd);
      }
      if (e3 != vEnd) {
        auto pos = atomicAdd(&size, 1);
        eArr1[pos] = e1;
        eArr2[pos] = e2;
        eArr3[pos] = e3;
      }
      __syncthreads();
    }
  }
  __syncthreads();
  for (EdgeT j = threadIdx.x; j < size; j += blockDim.x) {
    EdgeT ee1 = eArr1[j];
    EdgeT ee2 = edgesId[eArr2[j]];
    EdgeT ee3 = edgesId[eArr3[j]];
    PeelTriangle(level, inCurr, next, nextTail, inNext, edgesSup, processed, ee1, ee2, ee3);
  }
}

// 子任务循环迭代消减truss
void SubLevel(const EdgeT *nodeIndex, const NodeT *adj, const EdgeT *curr, bool *inCurr, EdgeT *currTail,
              NodeT *edgesSup, NodeT level, EdgeT *next, bool *inNext, EdgeT *nextTail, bool *processed,
              const EdgeT *edgesId, const uint64_t *halfEdges) {
  SubLevelKernel<<<*currTail, BLOCK_SIZE, BLOCK_SIZE * sizeof(EdgeT) * 2 * 3>>>(
      nodeIndex, adj, curr, inCurr, *currTail, edgesSup, level, next, inNext, nextTail, processed, edgesId, halfEdges);
  CUDA_TRY(hipDeviceSynchronize());

  UpdateProcessKernel<<<DIV_ROUND_UP(*currTail, BLOCK_SIZE), BLOCK_SIZE>>>(curr, *currTail, inCurr, processed);
  CUDA_TRY(hipDeviceSynchronize());
}

// 获取各层次truss的边的数量
NodeT DisplayStats(const NodeT *edgesSup, EdgeT halfEdgesNum, NodeT minK) {
  thrust::device_ptr<NodeT> edgesSupPtr(const_cast<NodeT *>(edgesSup));
  NodeT maxSup = *thrust::max_element(edgesSupPtr, edgesSupPtr + halfEdgesNum);

  EdgeT numEdgesWithMaxSup = thrust::count(edgesSupPtr, edgesSupPtr + halfEdgesNum, maxSup);

  log_info("Max-truss: %u  Edges in Max-truss: %u", maxSup + 2, numEdgesWithMaxSup);
  if (maxSup + 2 >= minK) {
    printf("kmax = %u, Edges in kmax-truss = %u.\n", maxSup + 2, numEdgesWithMaxSup);
  }
  return maxSup + 2;
}

void InitCuda(EdgeT *&currTail, EdgeT *&nextTail, bool *&processed, bool *&inCurr, bool *&inNext, EdgeT *&curr,
              EdgeT *&next, const EdgeT halfEdgesNum) {
  CUDA_TRY(hipMallocManaged((void **)&currTail, sizeof(EdgeT)));
  CUDA_TRY(hipMallocManaged((void **)&nextTail, sizeof(EdgeT)));
  *currTail = 0;
  *nextTail = 0;

  CUDA_TRY(hipMallocManaged((void **)&processed, halfEdgesNum * sizeof(bool)));
  CUDA_TRY(hipMallocManaged((void **)&inCurr, halfEdgesNum * sizeof(bool)));
  CUDA_TRY(hipMallocManaged((void **)&inNext, halfEdgesNum * sizeof(bool)));
  CUDA_TRY(hipMallocManaged((void **)&curr, halfEdgesNum * sizeof(EdgeT)));
  CUDA_TRY(hipMallocManaged((void **)&next, halfEdgesNum * sizeof(EdgeT)));

  CUDA_TRY(hipMemset(processed, 0, halfEdgesNum * sizeof(bool)));
  CUDA_TRY(hipMemset(inCurr, 0, halfEdgesNum * sizeof(bool)));
  CUDA_TRY(hipMemset(inNext, 0, halfEdgesNum * sizeof(bool)));
}

__global__ void DetectDeletedEdgesKernel(EdgeT *nodeIndex, EdgeT *edgesId, bool *processed, NodeT nodesNum,
                                         EdgeT *newOffsets, bool *edgesDeleted) {
  __shared__ NodeT cnts[WARPS_PER_BLOCK];

  auto gtid = threadIdx.x + blockIdx.x * blockDim.x;
  auto gtnum = blockDim.x * gridDim.x;
  auto gwid = gtid >> WARP_BITS;
  auto gwnum = gtnum >> WARP_BITS;
  auto lane = threadIdx.x & WARP_MASK;
  auto lwid = threadIdx.x >> WARP_BITS;

  for (auto u = gwid; u < nodesNum; u += gwnum) {
    if (0 == lane) {
      cnts[lwid] = 0;
    }
    __syncwarp();

    auto start = nodeIndex[u];
    auto end = nodeIndex[u + 1];
    for (auto v_idx = start + lane; v_idx < end; v_idx += WARP_SIZE) {
      auto target_edge_idx = edgesId[v_idx];
      edgesDeleted[v_idx] = !processed[target_edge_idx];
      if (edgesDeleted[v_idx]) {
        atomicAdd(&cnts[lwid], 1);
      }
    }
    __syncwarp();

    if (0 == lane) {
      newOffsets[u] = cnts[lwid];
    }
  }
}

template <typename T>
struct IsDelete : public thrust::unary_function<T, bool> {
  template <typename Tuple>
  __host__ __device__ bool operator()(const Tuple &tuple) {
    auto y = thrust::get<1>(tuple);
    return y;
  }
};

void ShrinkGraph(EdgeT *&nodeIndex, NodeT *&adj, EdgeT *&edgesId, bool *&processed, NodeT nodesNum,
                 EdgeT *&newNodeIndex, NodeT *&newAdj, EdgeT *&newEdgesId, bool *&edgesDeleted, EdgeT oldEdgesNum,
                 EdgeT newEdgesNum) {
  DetectDeletedEdgesKernel<<<GRID_SIZE, BLOCK_SIZE>>>(nodeIndex, edgesId, processed, nodesNum, newNodeIndex,
                                                      edgesDeleted);

  thrust::device_ptr<EdgeT> newNodeIndexPtr(newNodeIndex);
  thrust::exclusive_scan(newNodeIndexPtr, newNodeIndexPtr + nodesNum + 1, newNodeIndexPtr);

  swap(nodeIndex, newNodeIndex);

  thrust::device_ptr<NodeT> adjPtr(adj);
  thrust::device_ptr<NodeT> newAdjPtr(newAdj);
  thrust::device_ptr<EdgeT> edgesIdPtr(edgesId);
  thrust::device_ptr<EdgeT> newEdgesIdPtr(newEdgesId);
  thrust::device_ptr<bool> deleteEdgesPtr(edgesDeleted);

  thrust::copy_if(
      thrust::make_zip_iterator(thrust::make_tuple(adjPtr, deleteEdgesPtr)),
      thrust::make_zip_iterator(thrust::make_tuple(adjPtr + oldEdgesNum * 2, deleteEdgesPtr + oldEdgesNum * 2)),
      thrust::make_zip_iterator(thrust::make_tuple(newAdjPtr, thrust::make_discard_iterator())),
      IsDelete<decltype(thrust::make_tuple(adjPtr, deleteEdgesPtr))>());

  thrust::copy_if(
      thrust::make_zip_iterator(thrust::make_tuple(edgesIdPtr, deleteEdgesPtr)),
      thrust::make_zip_iterator(thrust::make_tuple(edgesIdPtr + oldEdgesNum * 2, deleteEdgesPtr + oldEdgesNum * 2)),
      thrust::make_zip_iterator(thrust::make_tuple(newEdgesIdPtr, thrust::make_discard_iterator())),
      IsDelete<decltype(thrust::make_tuple(edgesIdPtr, deleteEdgesPtr))>());

  swap(adj, newAdj);
  swap(edgesId, newEdgesId);
}

// 求解k-truss的主流程
void KTruss(EdgeT *nodeIndex, NodeT *adj, EdgeT *edgesId, NodeT nodesNum, const uint64_t *halfEdges, EdgeT halfEdgesNum,
            NodeT *edgesSup, NodeT startLevel) {
  EdgeT *currTail;
  EdgeT *nextTail;

  bool *processed;
  bool *inCurr;
  bool *inNext;
  EdgeT *curr;
  EdgeT *next;

  InitCuda(currTail, nextTail, processed, inCurr, inNext, curr, next, halfEdgesNum);

  NodeT *newAdj;
  EdgeT *newEdgesId;
  EdgeT *newNodeIndex;
  bool *edgesDeleted;

  CUDA_TRY(hipMallocManaged((void **)&newAdj, halfEdgesNum * 2 * sizeof(NodeT)));
  CUDA_TRY(hipMallocManaged((void **)&newEdgesId, halfEdgesNum * 2 * sizeof(EdgeT)));
  CUDA_TRY(hipMallocManaged((void **)&newNodeIndex, (nodesNum + 1) * sizeof(EdgeT)));
  CUDA_TRY(hipMallocManaged((void **)&edgesDeleted, halfEdgesNum * 2 * sizeof(bool)));

  NodeT level = startLevel;
  EdgeT todo = halfEdgesNum;
  EdgeT oriHalfEdgesNum = halfEdgesNum;
  EdgeT deleteEdgesNum = 0;
  if (level > 0u) {
    --level;
    ScanLessThanLevelKernel<<<DIV_ROUND_UP(oriHalfEdgesNum, BLOCK_SIZE), BLOCK_SIZE>>>(oriHalfEdgesNum, edgesSup, level,
                                                                                       curr, currTail, inCurr);
    CUDA_TRY(hipDeviceSynchronize());
    log_debug("level: %u currTail: %u restEdges: %u", level, *currTail, todo);

    while (*currTail > 0) {
      if ((deleteEdgesNum * 1.0 / oriHalfEdgesNum) > 0.05) {
        log_debug("ShrinkGraph: %u %u", deleteEdgesNum, todo);
        ShrinkGraph(nodeIndex, adj, edgesId, processed, nodesNum, newNodeIndex, newAdj, newEdgesId, edgesDeleted,
                    halfEdgesNum, todo);
        halfEdgesNum = todo;
        deleteEdgesNum = 0;
      }
      todo = todo - *currTail;

      deleteEdgesNum += *currTail;

      SubLevel(nodeIndex, adj, curr, inCurr, currTail, edgesSup, level, next, inNext, nextTail, processed, edgesId,
               halfEdges);

      std::swap(curr, next);
      std::swap(inCurr, inNext);

      *currTail = *nextTail;
      *nextTail = 0;

      log_debug("level: %u currTail: %u restEdges: %u deleteEdgesNum: %u", level, *currTail, todo, deleteEdgesNum);
    }
    ++level;
  } else {
    thrust::device_ptr<EdgeT> edgesSupPtr(edgesSup);
    level = *thrust::min_element(edgesSupPtr, edgesSupPtr + halfEdgesNum);
  }

  while (todo > 0) {
    //    if ((deleteEdgesNum * 1.0 / oriHalfEdgesNum) > 0.05) {
    //      log_debug("ShrinkGraph: %u %u", deleteEdgesNum, todo);
    //      ShrinkGraph(nodeIndex, adj, edgesId, processed, nodesNum, newNodeIndex, newAdj, newEdgesId, edgesDeleted,
    //                  halfEdgesNum, todo);
    //      halfEdgesNum = todo;
    //      deleteEdgesNum = 0;
    //    }

    ScanKernel<<<DIV_ROUND_UP(oriHalfEdgesNum, BLOCK_SIZE), BLOCK_SIZE>>>(oriHalfEdgesNum, edgesSup, level, curr,
                                                                          currTail, inCurr);
    CUDA_TRY(hipDeviceSynchronize());
    log_debug("level: %u currTail: %u restEdges: %u", level, *currTail, todo);

    while (*currTail > 0) {
      if ((deleteEdgesNum * 1.0 / oriHalfEdgesNum) > 0.1) {
        log_debug("ShrinkGraph: %u %u", deleteEdgesNum, todo);
        ShrinkGraph(nodeIndex, adj, edgesId, processed, nodesNum, newNodeIndex, newAdj, newEdgesId, edgesDeleted,
                    halfEdgesNum, todo);
        halfEdgesNum = todo;
        deleteEdgesNum = 0;
      }

      todo = todo - *currTail;

      deleteEdgesNum += *currTail;

      SubLevel(nodeIndex, adj, curr, inCurr, currTail, edgesSup, level, next, inNext, nextTail, processed, edgesId,
               halfEdges);

      std::swap(curr, next);
      std::swap(inCurr, inNext);

      *currTail = *nextTail;
      *nextTail = 0;

      log_debug("level: %u currTail: %u restEdges: %u deleteEdgesNum: %u", level, *currTail, todo, deleteEdgesNum);
    }
    ++level;
  }

  CUDA_TRY(hipFree(currTail));
  CUDA_TRY(hipFree(nextTail));
  CUDA_TRY(hipFree(processed));
  CUDA_TRY(hipFree(inCurr));
  CUDA_TRY(hipFree(inNext));
  CUDA_TRY(hipFree(curr));
  CUDA_TRY(hipFree(next));
}
